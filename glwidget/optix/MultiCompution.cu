#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include "volume.cuh"
rtDeclareVariable(uint, gridIndex, rtLaunchIndex, );
//rtDeclareVariable(int, maxIterator, , );
rtDeclareVariable(int, curIterator, , );
rtDeclareVariable(float, ee, , );
rtDeclareVariable(float, J_mean, , );
rtDeclareVariable(float, dx, , );
rtDeclareVariable(float, weight, , );
rtBuffer<float3, 1>    gridFluence;

static __device__ __inline__ float3  GetSigmaT0( int i)	
{
	return volume_density[i]*sigma_t*make_float3(1.f);
}
static __device__ __inline__ float3  GetSigmaT1( int i)	
{
	return max(volume_density[i]*sigma_t, 1.f/pow(10.f, 3)/20.f)*make_float3(1.f);
}

static __device__ __inline__ float3  max( float3 a, float3 b)
{
	float3 ret;
	ret.x = max(a.x, b.x);
	ret.y = max(a.y, b.z);
	ret.z = max(a.y, b.z);
	return ret;
}
//static __device__ __inline__ bool compare(int3 a, int3 b)
//{
//
//}

static __device__ __inline__ bool safe_index(int3& xyz,  int addx, int addy, int addz)
{
	xyz += make_int3(addx, addy, addz);
	int3 minIndex = make_int3(0);
	int3 maxIndex = make_int3(index_x-1, index_y-1, index_z-1);
	if (xyz.x>index_x-1 || xyz.x<0)
		return 0;	
	if (xyz.y>index_y-1 || xyz.y<0)
		return 0;	
	if (xyz.z>index_z-1 || xyz.z<0)
		return 0;	
	if (min(xyz, minIndex)!= minIndex) return 0;
	if (max(xyz, maxIndex)!= maxIndex) return 0;
	return 1;
}

static __device__ __inline__ int safe_index(int i,  int addx, int addy, int addz)
{
	int3 xyz = i2xyz(i);
	if (safe_index(xyz, addx, addy, addz))
		return xyz2i(xyz);
	return -1;
}

static __device__ __inline__ float3 safeGetFlue(  int i,  int addx, int addy, int addz)
{
	int indexI =  safe_index( i, addx, addy, addz);
	if (indexI>0)
		return gridFluence[ indexI];
	return make_float3(0.f);
}



static __device__ __inline__ float3 D_flu(int indexI)
{
	
	float3 fluence10 = safeGetFlue( indexI, -1, 0,  0);
	float3 fluence11 = safeGetFlue( indexI, 1,  0,  0);
	float3 fluence20 = safeGetFlue( indexI, 0, -1,  0);
	float3 fluence21 = safeGetFlue( indexI, 0,  1,  0);
	float3 fluence30 = safeGetFlue( indexI, 0,  0, -1);
	float3 fluence31 = safeGetFlue( indexI, 0,  0,  1);
	float3 ret;
	float3 a1 = fluence11 -fluence10;
	float3 a2 = fluence21 -fluence20;
	float3 a3 = fluence31 -fluence30;

	ret.x = length(make_float3(a1.x, a2.x, a3.x)/2.f/dx);
	ret.y = length(make_float3(a1.y, a2.y, a3.y)/2.f/dx);
	ret.z = length(make_float3(a1.z, a2.z, a3.z)/2.f/dx);
	return ret;
}

static __device__ __inline__ float3 GetRp( int i)
{
	float3 x1 = max(D_flu(i), ee*J_mean*make_float3(1.f));
	float3 sigmaP = GetSigmaT0( i);
	float3 x2 = max(sigmaP*gridFluence[i], ee*J_mean*make_float3(1.f));
	return x1/x2;
}

static __device__ __inline__ float3 GetFr( float3 rp)
{
	float3 ret;
	ret.x = 2.f/(3.f + sqrt( 9.f+4.f*pow( rp.x, 2.f)));
	ret.y = 2.f/(3.f + sqrt( 9.f+4.f*pow( rp.y, 2.f)));
	ret.z = 2.f/(3.f + sqrt( 9.f+4.f*pow( rp.z, 2.f)));
	return ret;
}

static __device__ __inline__ float3 GetDp( int i)
{
	float3 sigmaP = GetSigmaT1( i);
	float3 rp = GetRp( i);
	float3 fr = GetFr( rp);
	return fr/sigmaP;
}

static __device__ __inline__ float3 safeGetDp(  int i,  int addx, int addy, int addz)
{
	int indexI =  safe_index( i, addx, addy, addz);
	if (indexI>0)
		return GetDp( indexI);
	return make_float3(0.f);
}

//-----------------------------------------------------------------------------
//
//  Main program
//
//-----------------------------------------------------------------------------
RT_PROGRAM void MultiCompution()
{
	if (curIterator==1)
	{
		gridFluence[ gridIndex] = make_float3(1.f)*ee*dx;
		return;
	}
	int i = gridIndex;
	float3 update_fluence = make_float3(0.f);
	int3 xyzIndex = i2xyz( i);
	int tmpI = xyzIndex.x+xyzIndex.y+xyzIndex.z;
	if( tmpI%2 == curIterator%2) return;

	float3 sigmaP = GetSigmaT0( i);
	float3 Dp = GetDp( i);

	if ( 0)//volume_density[ i] == 0.f)
	{

	}
	else
	{
		float3 Ds10 = (safeGetDp(i, -1, 0, 0)+Dp)/2.f;
		float3 Ds11 = (safeGetDp(i,  1, 0, 0)+Dp)/2.f;
		
		float3 Ds20 = (safeGetDp(i, 0, -1, 0)+Dp)/2.f;
		float3 Ds21 = (safeGetDp(i, 0,  1, 0)+Dp)/2.f;
		
		float3 Ds30 = (safeGetDp(i, 0, 0, -1)+Dp)/2.f;
		float3 Ds31 = (safeGetDp(i, 0, 0,  1)+Dp)/2.f;
		
		float3 fluence10 = safeGetFlue(  i, -1, 0,  0);
		float3 fluence11 = safeGetFlue(  i, 1,  0,  0);
		float3 fluence20 = safeGetFlue(  i, 0, -1,  0);
		float3 fluence21 = safeGetFlue(  i, 0,  1,  0);
		float3 fluence30 = safeGetFlue(  i, 0,  0, -1);
		float3 fluence31 = safeGetFlue(  i, 0,  0,  1);

		float3 numerator = dx*dx*gridBuffer[i]+(Ds10*fluence10 + Ds11*fluence11 + Ds20*fluence20+Ds21*fluence21+Ds30*fluence30+Ds31*fluence31);
		float3 denominator = (1.f-alpha_value)*sigmaP*dx*dx+(Ds10+Ds11+Ds20+Ds21+Ds30+Ds31);
		//error_p = (numerator - $fluence[i]*denominator)/$dx/$dx
		update_fluence = numerator/denominator;
	}
	gridFluence[i] = max(weight*update_fluence+(1.f-weight)* gridFluence[i], make_float3(0.f,0.f,0.f));
}

//-----------------------------------------------------------------------------
//
//  Exception program
//
//-----------------------------------------------------------------------------

RT_PROGRAM void exception()
{
	gridFluence[ gridIndex] =  make_float3(0.f,1.f,0.f);//bad_color;
}


