#include "hip/hip_runtime.h"
#include "ray.cuh"
// -----------------------------------------------------------------------------
//-----------------------------------------------------------------------------
//
// Mirror program
//
//-----------------------------------------------------------------------------



RT_PROGRAM void mirror_fresnel()
{
  float3 world_shading_normal   = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
  float3 world_geometric_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );

  float3 ffnormal = faceforward( world_shading_normal, -ray.direction, world_geometric_normal );

  float3 hitpoint = ray.origin + t_hit * ray.direction;
  current_prd.origin = hitpoint;
  current_prd.countEmitted = false;



  float rand_reflect = rnd(current_prd.seed);
  // check for external or internal reflection

  current_prd.direction = reflect(ray.direction, ffnormal);	

  current_prd.radiance = make_float3(0.0f,0.f,0.f);
}


RT_PROGRAM void shadow()
{
	current_prd_shadow.attenuation = make_float3(0.f);
	rtTerminateRay();
}