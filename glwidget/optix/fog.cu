#include "hip/hip_runtime.h"
//#include "ray.cuh"
//#include "math.cuh"
#include "volume.cuh"
// -----------------------------------------------------------------------------
//-----------------------------------------------------------------------------
//
// Fog program
//
//-----------------------------------------------------------------------------

rtDeclareVariable(float3,        glass_color, , );
rtDeclareVariable(float,         index_of_refraction, , );
rtDeclareVariable(float,        fresnel_exponent, , );
rtDeclareVariable(float,        fresnel_minimum, , );
rtDeclareVariable(float,        fresnel_maximum, , );


static __device__ __inline__ float woodcockTracking( const Ray& current_ray, float maxLength, float maxExtinction)
{
	float3 p = current_ray.origin;
	float3 dir = current_ray.direction;
	if (maxExtinction==0.f)
		return 100000000.f;

	float r = rnd(current_prd.seed);
	float d=-log(1.f-r)/maxExtinction;
	//*for inhomogeneous media
	int itimes = 0;
	float sigma;
	while (d < maxLength)
	{
		itimes++;
		if (itimes>300) break;
		sigma=maxExtinction*get_density(p+d*dir);
		r = rnd(current_prd.seed);
		if (r<sigma/maxExtinction)
			break;
		r = rnd(current_prd.seed);
		d-=log(1.f-r)/maxExtinction;
	}
	return d;
}

static __device__ __inline__ float woodcockTracking_shadow( const Ray& current_ray, float maxLength, float maxExtinction)
{
	float3 p = current_ray.origin;
	float3 dir = current_ray.direction;
	if (maxExtinction==0.f)
		return 100000000.f;

	float r = rnd(current_prd_shadow.seed);
	float d=-log(1.f-r)/maxExtinction;
	//*for inhomogeneous media
	int itimes = 0;
	float sigma;
	while (d < maxLength)
	{
		itimes++;
		if (itimes>300) break;
		sigma=maxExtinction*get_density(p+d*dir);
		r = rnd(current_prd_shadow.seed);
		if (r<sigma/maxExtinction)
			break;
		r = rnd(current_prd_shadow.seed);
		d-=log(1.f-r)/maxExtinction;
	}
	return d;
}


RT_PROGRAM void fog_shadow()
{
	float maxLength = 500.f;//ray.tmax;
	float d = woodcockTracking_shadow( ray, maxLength, sigma_t);
	if(d< maxLength-scene_epsilon)
	{
		current_prd_shadow.attenuation = make_float3(0.f);
	}
	else
	{
		current_prd_shadow.attenuation = make_float3(1.f);
	}
	//current_prd_shadow.attenuation = make_float3(0.f, 0.f, 1.f);
	rtTerminateRay();
}



RT_PROGRAM void fog__closest_hit_radiance()
{
	float3 result = make_float3(0.0f);
	float3 world_shading_normal   = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
	float3 world_geometric_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );

	float3 ffnormal = faceforward( world_shading_normal, -ray.direction, world_geometric_normal );

	float3 hitpoint = ray.origin + t_hit * ray.direction;
	//current_prd.origin = hitpoint;
	current_prd.countEmitted = false;

	float iof;
	if (current_prd.inside) {
		// Shoot outgoing ray
		iof = 1.0f/index_of_refraction;
	} 
	else 
	{
		iof = index_of_refraction;
	}

	if (current_prd.inside) 
	{
		float d = woodcockTracking( ray, t_hit, sigma_t);//1000.f;//woodcockTracking(0.1f, r1);
		if (d>t_hit)
		{
			current_prd.origin = ray.origin + (t_hit+scene_epsilon) * ray.direction;//hitpoint;
			refract(current_prd.direction, ray.direction, ffnormal, iof);
			current_prd.inside = !current_prd.inside;
		}
		else
		{
			current_prd.origin = ray.origin+d*ray.direction;
			float z1=rnd(current_prd.seed);
			float z2=rnd(current_prd.seed);
			float3 p;
			cosine_sample_hemisphere(z1, z2, p);
			float3 v1, v2;
			createONB(ray.direction, v1, v2);
			current_prd.direction = v1 * p.x + v2 * p.y + ray.direction * p.z;
			current_prd.attenuation*=alpha_value;

			// Compute direct light...
			// Or shoot one...
			unsigned int num_lights = lights.size();

			for(int i = 0; i < num_lights; ++i) 
			{
				ParallelogramLight light = lights[i];
				float z1 = rnd(current_prd.seed);
				float z2 = rnd(current_prd.seed);
				float3 light_pos = light.corner + light.v1 * z1 + light.v2 * z2;

				float Ldist = length(light_pos - current_prd.origin);
				float3 L = normalize(light_pos - current_prd.origin);
				float nDl = dot( current_prd.direction, L );
				float LnDl = dot( light.normal, L );
				float A = length(cross(light.v1, light.v2));

				// cast shadow ray
				if ( nDl > 0.0f && LnDl > 0.0f ) 
				{
					PerRayData_pathtrace_shadow shadow_prd;
					shadow_prd.attenuation = make_float3(1.f);
					Ray shadow_ray = make_Ray( current_prd.origin, L, pathtrace_shadow_ray_type, scene_epsilon, Ldist );
					shadow_prd.origin = current_prd.origin;
					shadow_prd.direction = L;
					shadow_prd.seed = current_prd.seed;
					rtTrace(top_object, shadow_ray, shadow_prd);

					float3 light_attenuation = shadow_prd.attenuation;
					if(fmaxf(light_attenuation) > 0.0f)
					{
						float weight=nDl * LnDl * A / (M_PIf*Ldist*Ldist);
						result += light.emission * weight*light_attenuation;
					}
				}
			}
		}
	}
	else
	{
		current_prd.origin = hitpoint;
		float3 t;
		refract(t, ray.direction, ffnormal, iof);
		float rand_reflect = rnd(current_prd.seed);
		// check for external or internal reflection
		float cos_theta = dot(ray.direction, ffnormal);
		if (cos_theta < 0.0f)
			cos_theta = -cos_theta;
		else
			cos_theta = dot(t, ffnormal);

		float reflection = fresnel_schlick(cos_theta, fresnel_exponent, fresnel_minimum, fresnel_maximum);

		if (0)//(rand_reflect<reflection)
		{
			//reflect
			current_prd.direction = reflect(ray.direction, ffnormal);	
		}
		else
		{
			//refract
			refract(current_prd.direction, ray.direction, ffnormal, iof);
			current_prd.inside = !current_prd.inside;

		}
	}

	current_prd.radiance = result;

}


