#include "hip/hip_runtime.h"
//#include "ray.cuh"
//#include "math.cuh"
#include "volume.cuh"
// -----------------------------------------------------------------------------
//-----------------------------------------------------------------------------
//
// Fog program
//
//-----------------------------------------------------------------------------

rtDeclareVariable(float3,        glass_color, , );
rtDeclareVariable(float,         index_of_refraction, , );
rtDeclareVariable(float,        fresnel_exponent, , );
rtDeclareVariable(float,        fresnel_minimum, , );
rtDeclareVariable(float,        fresnel_maximum, , );

RT_PROGRAM void fog_shadow()
{

	float d=200.f;
	float3 p=current_prd_shadow.origin;
	float3 dir=current_prd_shadow.direction;
	float l=0.f;
	float step=30.f;
	float rand0=0.f;
	l+=rand0*step;
	float tau = 0.f;
	tau+=get_density(p)*rand0;
	while (l<=d)
	{
		tau+=get_density(p+dir*l);
		l+=step;
	}
	tau*=step;
	//return tau//*$sigma_t.x 
	current_prd_shadow.attenuation *= make_float3(0.f);//exp(sigma_t*tau);//make_float3(0.f);
	//rtTerminateRay();
}
	
static __device__ __inline__ float woodcockTracking2( PerRayData_pathtrace& current_ray, float maxLength, float maxExtinction)
{
  if (maxExtinction==0.f)
		return 100000000.f;
	float r1 = rnd(current_prd.seed);
	float d=-log(1.f-r1)/maxExtinction;

	float rand0 = rnd(current_prd.seed);
	float3 p=current_prd.origin;
	float3 v=normalize(current_prd.direction);
	float l=0.f;
	float span=3.f;
	l+=rand0*span;
	float len = 0.f;
	len+=get_density( p)*rand0;
	while (len*span<=d && l<=maxLength)
	{
		len+=get_density( p+v*l);
		l+=span;
	}
	float len2 = 0.f;
	float span2=span/20.f;
	if (len*span>d)
	{
		l-=span;
		len-=get_density( p+v*l);
		int tt=0;
		while (len*span+len2*span2<=d && tt<20)
		{
			tt++;
			len2+=get_density( p+v*l);
			l+=span2;
		}
	}
	return l;
}

RT_PROGRAM void fog__closest_hit_radiance()
{
	float3 result = make_float3(0.0f);
	float3 world_shading_normal   = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
	float3 world_geometric_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );
	
	float3 ffnormal = faceforward( world_shading_normal, -ray.direction, world_geometric_normal );
	
	float3 hitpoint = ray.origin + t_hit * ray.direction;
	//current_prd.origin = hitpoint;
	current_prd.countEmitted = false;
	
	float iof;
	if (current_prd.inside) {
		// Shoot outgoing ray
		iof = 1.0f/index_of_refraction;
	} 
	else 
	{
		iof = index_of_refraction;
	}
	
	if (current_prd.inside) 
	{
		float r1=rnd(current_prd.seed);
		float d = woodcockTracking2(current_prd, t_hit, sigma_t);//1000.f;//woodcockTracking(0.1f, r1);
		if (d>t_hit)
		{
			current_prd.origin = hitpoint;
			refract(current_prd.direction, ray.direction, ffnormal, iof);
			current_prd.inside = !current_prd.inside;
		}
		else
		{
			current_prd.origin = ray.origin+d*ray.direction;
			float z1=rnd(current_prd.seed);
			float z2=rnd(current_prd.seed);
			float3 p;
			cosine_sample_hemisphere(z1, z2, p);
			float3 v1, v2;
			createONB(ray.direction, v1, v2);
			current_prd.direction = v1 * p.x + v2 * p.y + ray.direction * p.z;
			current_prd.attenuation*=alpha_value;
			
			// Compute direct light...
			// Or shoot one...
			unsigned int num_lights = lights.size();
			
			for(int i = 0; i < num_lights; ++i) 
			{
				ParallelogramLight light = lights[i];
				float z1 = rnd(current_prd.seed);
				float z2 = rnd(current_prd.seed);
				float3 light_pos = light.corner + light.v1 * z1 + light.v2 * z2;
				
				float Ldist = length(light_pos - current_prd.origin);
				float3 L = normalize(light_pos - current_prd.origin);
				float nDl = dot( current_prd.direction, L );
				float LnDl = dot( light.normal, L );
				float A = length(cross(light.v1, light.v2));
				
				// cast shadow ray
				if ( nDl > 0.0f && LnDl > 0.0f ) 
				{
					PerRayData_pathtrace_shadow shadow_prd;
					shadow_prd.attenuation = make_float3(1.f);
					//Ray shadow_ray = make_Ray( hitpoint, L, pathtrace_shadow_ray_type, scene_epsilon, Ldist );
					shadow_prd.origin = current_prd.origin;
					shadow_prd.direction = L;
					//rtTrace(top_object, shadow_ray, shadow_prd);
					
					float3 p=shadow_prd.origin;
					float3 dir=shadow_prd.direction;
					float l=0.f;
					float step=3.f;
					float rand0=rnd(current_prd.seed);
					l+=rand0*step;
					float tau = 0.f;
					tau+=get_density(p)*rand0;
					while (l<=500.f)
					{
						tau+=get_density(p+dir*l);
						l+=step;
					}
					tau*=step;
					shadow_prd.attenuation = make_float3(1.f)* exp(-sigma_t*tau);//make_float3(0.f);
					
					float3 light_attenuation = shadow_prd.attenuation;
					if(fmaxf(light_attenuation) > 0.0f)
					{
						float weight=nDl * LnDl * A / (M_PIf*Ldist*Ldist);
						result += light.emission * weight*light_attenuation;
					}
				}
			}
		}
	}
	else
	{
		current_prd.origin = hitpoint;
		float3 t;
		refract(t, ray.direction, ffnormal, iof);
		float rand_reflect = rnd(current_prd.seed);
		// check for external or internal reflection
		float cos_theta = dot(ray.direction, ffnormal);
		if (cos_theta < 0.0f)
			cos_theta = -cos_theta;
		else
			cos_theta = dot(t, ffnormal);
		
		float reflection = fresnel_schlick(cos_theta, fresnel_exponent, fresnel_minimum, fresnel_maximum);
		
		if (0)//(rand_reflect<reflection)
		{
			//reflect
			current_prd.direction = reflect(ray.direction, ffnormal);	
		}
		else
		{
			//refract
			refract(current_prd.direction, ray.direction, ffnormal, iof);
			//current_prd.expand_rad = 3.f;
			//current_prd.attenuation*=20.f;//
			current_prd.inside = !current_prd.inside;
			
		}
	}
	
	
	
	//current_prd.attenuation*=5.f;
	current_prd.radiance = result;
	//current_prd.radiance = make_float3(0.f,0.f,0.f);
}