#include "hip/hip_runtime.h"
//#include "ray.cuh"
//#include "math.cuh"
#include "volume.cuh"
// -----------------------------------------------------------------------------
//-----------------------------------------------------------------------------
//
// Fog program
//
//-----------------------------------------------------------------------------

rtDeclareVariable(float3,        glass_color, , );
rtDeclareVariable(float,         index_of_refraction, , );
rtDeclareVariable(float,        fresnel_exponent, , );
rtDeclareVariable(float,        fresnel_minimum, , );
rtDeclareVariable(float,        fresnel_maximum, , );
rtDeclareVariable(float,        isRayMarching, , );
rtDeclareVariable(float,        isPreCompution, , );



static __device__ __inline__ float woodcockTracking( const Ray& current_ray, float maxLength, float maxExtinction)
{
	float3 p = current_ray.origin;
	float3 dir = current_ray.direction;
	if (maxExtinction==0.f)
		return 100000000.f;

	float r = rnd(current_prd.seed);
	float d=-log(1.f-r)/maxExtinction;
	//*for inhomogeneous media
	int itimes = 0;
	float sigma;
	while (d < maxLength)
	{
		itimes++;
		if (itimes>300) break;
		sigma=maxExtinction*get_density(p+d*dir);
		r = rnd(current_prd.seed);
		if (r<sigma/maxExtinction)
			break;
		r = rnd(current_prd.seed);
		d-=log(1.f-r)/maxExtinction;
	}
	return d;
}

static __device__ __inline__ float woodcockTracking_shadow( const Ray& current_ray, float maxLength, float maxExtinction)
{
	float3 p = current_ray.origin;
	float3 dir = current_ray.direction;
	if (maxExtinction==0.f)
		return 100000000.f;

	float r = rnd(current_prd_shadow.seed);
	float d=-log(1.f-r)/maxExtinction;
	//*for inhomogeneous media
	int itimes = 0;
	float sigma;
	while (d < maxLength)
	{
		itimes++;
		if (itimes>300) break;
		sigma=maxExtinction*get_density(p+d*dir);
		r = rnd(current_prd_shadow.seed);
		if (r<sigma/maxExtinction)
			break;
		r = rnd(current_prd_shadow.seed);
		d-=log(1.f-r)/maxExtinction;
	}
	return d;
}


RT_PROGRAM void fog_shadow()
{
	float maxLength = 20.f;//ray.tmax;
	float d = woodcockTracking_shadow( ray, maxLength, sigma_t);
	if(d< maxLength-scene_epsilon)
	{
		current_prd_shadow.attenuation = make_float3(0.f);
	}
	else
	{
		current_prd_shadow.attenuation = make_float3(1.f);
	}
	//current_prd_shadow.attenuation = make_float3(0.f, 0.f, 1.f);
	rtTerminateRay();
}



RT_PROGRAM void fog__closest_hit_radiance()
{
	float3 result = make_float3(0.0f);
	float3 world_shading_normal   = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
	float3 world_geometric_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );

	float3 ffnormal = faceforward( world_shading_normal, -ray.direction, world_geometric_normal );

	float3 hitpoint = ray.origin + t_hit * ray.direction;
	//current_prd.origin = hitpoint;
	current_prd.countEmitted = false;

	float iof;
	if (current_prd.inside) {
		// Shoot outgoing ray
		iof = 1.0f/index_of_refraction;
	} 
	else 
	{
		iof = index_of_refraction;
	}

	if (current_prd.inside) 
	{
		float d = woodcockTracking( ray, t_hit, sigma_t);//1000.f;//woodcockTracking(0.1f, r1);
		if (d>=t_hit)
		{
			current_prd.origin = hitpoint;
			refract(current_prd.direction, ray.direction, ffnormal, iof);
			current_prd.inside = !current_prd.inside;
		}
		else
		{
			if(isPreCompution<0.5f)
			{
				current_prd.origin = ray.origin+d*ray.direction;
			}
			else
			{
				current_prd.origin = ray.origin;
			}

			//////////////////////////////////////////////////////////////////
			//rtTerminateRay();
			if( isRayMarching>0.5f)
			{
				current_prd.done = true;
				current_prd.radiance = interpolation( current_prd.origin);
				current_prd.attenuation = make_float3(1.f);
				return;
			}
			float z1=rnd(current_prd.seed);
			float z2=rnd(current_prd.seed);
			//float3 p;
			//cosine_sample_hemisphere(z1, z2, p);
			//float3 v1, v2;
			//createONB(ray.direction, v1, v2);
			//current_prd.direction = v1 * p.x + v2 * p.y + ray.direction * p.z;
			current_prd.direction = SampleHG( z1, z2, ray.direction);
			current_prd.attenuation*=alpha_value;

			// Compute direct light...
			// Or shoot one...
			unsigned int num_lights = lights.size();

			for(int i = 0; i < num_lights; ++i) 
			{
				if(isPreCompution<0.5f&&isSingle>0.5f&&current_prd.depth>=2)
					break;
				if(isPreCompution>0.5f&&current_prd.depth>=1)
					break;
				ParallelogramLight light = lights[i];
				float z1 = rnd(current_prd.seed);
				float z2 = rnd(current_prd.seed);
				float3 light_pos = light.corner + light.v1 * z1 + light.v2 * z2;

				float Ldist = length(light_pos - current_prd.origin);
				float3 L = normalize(light_pos - current_prd.origin);
				float nDl = 1.f;//dot( ray.direction, L );
				//if(isPreCompution>0.5f) nDl = 1.f;
				float LnDl = dot( light.normal, L );
				float A = length(cross(light.v1, light.v2));


				// cast shadow ray
				if ( nDl > 0.0f && LnDl > 0.0f ) 
				{
					PerRayData_pathtrace_shadow shadow_prd;
					shadow_prd.attenuation = make_float3(1.f);
					Ray shadow_ray = make_Ray( current_prd.origin, L, pathtrace_shadow_ray_type, scene_epsilon, Ldist );
					shadow_prd.origin = current_prd.origin;
					shadow_prd.direction = L;
					shadow_prd.seed = current_prd.seed;
					rtTrace(top_object, shadow_ray, shadow_prd);

					float3 light_attenuation = shadow_prd.attenuation;
					if(fmaxf(light_attenuation) > 0.0f)
					{
						float weight=nDl * LnDl * A / (M_PIf*Ldist*Ldist);
						result += light.emission * weight*light_attenuation;
					}
				}
			}
			if(isPreCompution>0.5f)
			{
				current_prd.origin = ray.origin+d*ray.direction;
			}
			//if(isPreCompution<0.5f&&isSingle>0.5f&&current_prd.depth>=3)
			//{
			//	current_prd.done = true;
			//}
			
		}
	}
	else
	{
		current_prd.origin = hitpoint;
		float3 t;
		refract(t, ray.direction, ffnormal, iof);
		float rand_reflect = rnd(current_prd.seed);
		// check for external or internal reflection
		float cos_theta = dot(ray.direction, ffnormal);
		if (cos_theta < 0.0f)
			cos_theta = -cos_theta;
		else
			cos_theta = dot(t, ffnormal);

		float reflection = fresnel_schlick(cos_theta, fresnel_exponent, fresnel_minimum, fresnel_maximum);

		if (0)//(rand_reflect<reflection)
		{
			//reflect
			current_prd.direction = reflect(ray.direction, ffnormal);	
		}
		else
		{
			//refract
			refract(current_prd.direction, ray.direction, ffnormal, iof);
			current_prd.inside = !current_prd.inside;

		}
	}

	current_prd.radiance = result;

}


