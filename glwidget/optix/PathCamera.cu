#include "hip/hip_runtime.h"
#include "ray.cuh"
#include "envmap.cuh"
rtDeclareVariable(uint2,      launch_index, rtLaunchIndex, );

rtDeclareVariable(float3,        eye, , );
rtDeclareVariable(float3,        U, , );
rtDeclareVariable(float3,        V, , );
rtDeclareVariable(float3,        W, , );
rtDeclareVariable(unsigned int,  frame_number, , );
rtDeclareVariable(unsigned int,  sqrt_num_samples, , );
rtBuffer<float4, 2>              output_buffer;
rtDeclareVariable(float,  hasBackground, , );
rtDeclareVariable(float,  radianceMultipler, , );
//rtDeclareVariable(float,  hasHDR, , );


//-----------------------------------------------------------------------------
//
//  Camera program -- main ray tracing loop
//
//-----------------------------------------------------------------------------

RT_PROGRAM void pathtrace_camera()
{
	size_t2 screen = output_buffer.size();

	float2 inv_screen = 1.0f/make_float2(screen) * 2.f;
	float2 pixel = (make_float2(launch_index)) * inv_screen - 1.f;

	float2 jitter_scale = inv_screen / sqrt_num_samples;
	unsigned int samples_per_pixel = sqrt_num_samples*sqrt_num_samples;
	float3 result = make_float3(0.0f);

	unsigned int seed = tea<16>(screen.x*launch_index.y+launch_index.x, frame_number);
	do {
		unsigned int x = samples_per_pixel%sqrt_num_samples;
		unsigned int y = samples_per_pixel/sqrt_num_samples;
		float2 jitter = make_float2(x-rnd(seed), y-rnd(seed));
		float2 d = pixel + jitter*jitter_scale;
		float3 ray_origin = eye;
		float3 ray_direction = normalize(d.x*U + d.y*V + W);

		PerRayData_pathtrace prd;
		prd.result = make_float3(0.f);
		prd.attenuation = make_float3(1.f);
		prd.countEmitted = true;
		prd.done = false;
		prd.inside = false;
		prd.seed = seed;
		prd.depth = 0;

		for(;;) {
			Ray ray = make_Ray(ray_origin, ray_direction, pathtrace_ray_type, scene_epsilon, RT_DEFAULT_MAX);
			rtTrace(top_object, ray, prd);
			if(prd.done ||(prd.depth >= max_depth)) {
				prd.result += prd.radiance * prd.attenuation;
				break;
			}

			// RR
			if(prd.depth >= rr_begin_depth){
				float pcont = fmaxf(prd.attenuation);
				if(rnd(prd.seed) >= pcont)
					break;
				prd.attenuation /= pcont;
			}
			prd.depth++;
			prd.result += prd.radiance * prd.attenuation;
			ray_origin = prd.origin;
			ray_direction = prd.direction;
		} // eye ray

		result += prd.result;
		seed = prd.seed;
	} while (--samples_per_pixel);

	float3 pixel_color = result/(sqrt_num_samples*sqrt_num_samples);
	/*
	if(launch_index.x==0&&launch_index.y==0)
	{
	rtPrintf("[%f,%f,%f]",pixel_color.x,pixel_color.y,pixel_color.z);
	}*/

	pixel_color *= radianceMultipler;
	if (frame_number > 1)
	{
		float a = 1.0f / (float)frame_number;
		float b = ((float)frame_number - 1.0f) * a;
		float3 old_color = make_float3(output_buffer[launch_index]);
		output_buffer[launch_index] = make_float4(a * pixel_color + b * old_color, 0.0f);
	}
	else
	{
		output_buffer[launch_index] = make_float4(pixel_color, 0.0f);
	}
}


//-----------------------------------------------------------------------------
//
//  Exception program
//
//-----------------------------------------------------------------------------

RT_PROGRAM void exception()
{
	output_buffer[launch_index] = make_float4(bad_color, 0.0f);
}


//-----------------------------------------------------------------------------
//
//  Miss program
//
//-----------------------------------------------------------------------------

RT_PROGRAM void miss()
{
	current_prd.radiance = bg_color;
	current_prd.done = true;
}


RT_PROGRAM void envmap_miss()
{
	float theta = atan2f( ray.direction.x, ray.direction.z );
	float phi   = M_PIf * 0.5f -  acosf( ray.direction.y );
	float u     = (theta + M_PIf) * (0.5f * M_1_PIf);
	float v     = 0.5f * ( 1.0f + sin(phi) );
	current_prd.radiance = bg_color;
	if(hasBackground>0.5f || current_prd.depth>2)
	{
		current_prd.radiance = envmapEvalL(ray.direction);//make_float3( tex2D(envmap, u, v) )*1.f;
	}
	current_prd.done = true;
	//current_prd.attenuation *= 0.1f;
}
