#include "hip/hip_runtime.h"
#include "ray.cuh"

rtDeclareVariable(float3,        eye, , );
rtDeclareVariable(float3,        U, , );
rtDeclareVariable(float3,        V, , );
rtDeclareVariable(float3,        W, , );
rtDeclareVariable(float3,        bad_color, , );
rtDeclareVariable(unsigned int,  frame_number, , );
rtDeclareVariable(unsigned int,  sqrt_num_samples, , );
rtBuffer<float4, 2>              output_buffer;

//-----------------------------------------------------------------------------
//
//  Camera program -- main ray tracing loop
//
//-----------------------------------------------------------------------------

RT_PROGRAM void pathtrace_camera()
{
	size_t2 screen = output_buffer.size();

	float2 inv_screen = 1.0f/make_float2(screen) * 2.f;
	float2 pixel = (make_float2(launch_index)) * inv_screen - 1.f;

	float2 jitter_scale = inv_screen / sqrt_num_samples;
	unsigned int samples_per_pixel = sqrt_num_samples*sqrt_num_samples;
	float3 result = make_float3(0.0f);

	unsigned int seed = tea<16>(screen.x*launch_index.y+launch_index.x, frame_number);
	do {
		unsigned int x = samples_per_pixel%sqrt_num_samples;
		unsigned int y = samples_per_pixel/sqrt_num_samples;
		float2 jitter = make_float2(x-rnd(seed), y-rnd(seed));
		float2 d = pixel + jitter*jitter_scale;
		float3 ray_origin = eye;
		float3 ray_direction = normalize(d.x*U + d.y*V + W);

		PerRayData_pathtrace prd;
		prd.result = make_float3(0.f);
		prd.attenuation = make_float3(1.f);
		prd.countEmitted = true;
		prd.done = false;
		prd.inside = false;
		prd.seed = seed;
		prd.depth = 0;
		prd.expand_rad = 1.f;

		for(;;) {
			Ray ray = make_Ray(ray_origin, ray_direction, pathtrace_ray_type, scene_epsilon, RT_DEFAULT_MAX);
			rtTrace(top_object, ray, prd);
			if(prd.done ||(prd.depth >= max_depth)) {
				prd.result += prd.radiance * prd.attenuation*prd.expand_rad;
				break;
			}

			// RR
			if(prd.depth >= rr_begin_depth){
				float pcont = fmaxf(prd.attenuation);
				if(rnd(prd.seed) >= pcont)
					break;
				prd.attenuation /= pcont;
			}
			prd.depth++;
			prd.result += prd.radiance * prd.attenuation*prd.expand_rad;
			ray_origin = prd.origin;
			ray_direction = prd.direction;
		} // eye ray

		result += prd.result;
		seed = prd.seed;
	} while (--samples_per_pixel);

	float3 pixel_color = result/(sqrt_num_samples*sqrt_num_samples);
	/*
	if(launch_index.x==0&&launch_index.y==0)
	{
	rtPrintf("[%f,%f,%f]",pixel_color.x,pixel_color.y,pixel_color.z);
	}*/


	if (frame_number > 1)
	{
		float a = 1.0f / (float)frame_number;
		float b = ((float)frame_number - 1.0f) * a;
		float3 old_color = make_float3(output_buffer[launch_index]);
		output_buffer[launch_index] = make_float4(a * pixel_color + b * old_color, 0.0f);
	}
	else
	{
		output_buffer[launch_index] = make_float4(pixel_color, 0.0f);
	}
}


//-----------------------------------------------------------------------------
//
//  Exception program
//
//-----------------------------------------------------------------------------

RT_PROGRAM void exception()
{
	output_buffer[launch_index] = make_float4(bad_color, 0.0f);
}


//-----------------------------------------------------------------------------
//
//  Miss program
//
//-----------------------------------------------------------------------------

RT_PROGRAM void miss()
{
	current_prd.radiance = bg_color;
	current_prd.done = true;
}


rtTextureSampler<float4, 2> envmap;
RT_PROGRAM void envmap_miss()
{
  float theta = atan2f( ray.direction.x, ray.direction.z );
  float phi   = M_PIf * 0.5f -  acosf( ray.direction.y );
  float u     = (theta + M_PIf) * (0.5f * M_1_PIf);
  float v     = 0.5f * ( 1.0f + sin(phi) );
  current_prd.radiance = make_float3( tex2D(envmap, u, v) );
  current_prd.done = true;
  //current_prd.attenuation *= 0.1f;
}
