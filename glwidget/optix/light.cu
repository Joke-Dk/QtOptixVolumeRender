#include "hip/hip_runtime.h"
#include "ray.cuh"

rtDeclareVariable(float3,        emission_color, , );

RT_PROGRAM void diffuseEmitter()
{
  current_prd.radiance = current_prd.countEmitted? emission_color : make_float3(1.f);
  current_prd.done = true;
}

RT_PROGRAM void diffuseEmitterShadow()
{
	//rtTerminateRay();
}