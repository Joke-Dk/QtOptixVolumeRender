#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include "volume.cuh"
#include "MultiCompution.cuh"
#include "envmap.cuh"
rtDeclareVariable(uint, gridIndex, rtLaunchIndex, );
rtDeclareVariable(int, numSampling, , );
rtDeclareVariable(unsigned int,  numCompution, , );
//rtBuffer<float4, 3>    gridBuffer;
static __device__ __inline__ float3 GetPosition( int3 index)
{
	float3 p01 = P1-P0;
	return make_float3(p01.x/(float)(index_x-1)*(float)(index.x)+P0.x, p01.y/(float)(index_y-1)*(float)(index.y)+P0.y, p01.z/(float)(index_z-1)*(float)(index.z)+P0.z );
}

RT_PROGRAM void MultiCompution()
{
	if (curIterator==1)
	{
		//gridBuffer[gridIndex] *= GetDensity(gridIndex)*sigma_t;
		gridFluence[ gridIndex] = make_float3(1.f)*ee*dx*J_mean;
		return;
	}
	int i = gridIndex;
	float3 update_fluence = make_float3(0.f);
	int3 xyzIndex = i2xyz( i);
	int tmpI = xyzIndex.x+xyzIndex.y+xyzIndex.z;
	if( tmpI%2 == curIterator%2) return;

	float3 sigmaP = GetSigmaT0( i);
	float3 Dp = GetDp( i);

	if ( GetDensity( i) == 0.f)
	{
		//gridFluence[ gridIndex] = gridBuffer[i];
		//return;
		int countTmp = 0;
		for ( int i0=max( xyzIndex.x-1, 0); i0<=min(xyzIndex.x+1, index_x-1); ++i0)
			for (int j0=max( xyzIndex.y-1, 0); j0<=min(xyzIndex.y+1, index_y-1); ++j0)
				for (int k0=max( xyzIndex.z-1, 0); k0<=min(xyzIndex.z+1, index_z-1);++k0)
					if (GetDensity(xyz2i(i0, j0, k0))!=0.f && !(i0==xyzIndex.x && j0==xyzIndex.y && k0==xyzIndex.z))
					{
						countTmp++;
						float3 Ds = make_float3(1.f)/3.f/GetSigmaT2(i);//*2.5f;//2.5f
						update_fluence += 2.f*Ds*gridFluence[xyz2i( i0, j0, k0)]/(dx*make_float3(1.f)+2.f*Ds);
					}
		if(countTmp!=0)
			update_fluence /= float(countTmp);
		else
			return;
	}
	else
	{
		float3 Ds10 = (safeGetDp(i, -1, 0, 0)+Dp)/2.f;
		float3 Ds11 = (safeGetDp(i,  1, 0, 0)+Dp)/2.f;
		
		float3 Ds20 = (safeGetDp(i, 0, -1, 0)+Dp)/2.f;
		float3 Ds21 = (safeGetDp(i, 0,  1, 0)+Dp)/2.f;
		
		float3 Ds30 = (safeGetDp(i, 0, 0, -1)+Dp)/2.f;
		float3 Ds31 = (safeGetDp(i, 0, 0,  1)+Dp)/2.f;
		
		float3 fluence10 = safeGetFlue(  i, -1, 0,  0);
		float3 fluence11 = safeGetFlue(  i, 1,  0,  0);
		float3 fluence20 = safeGetFlue(  i, 0, -1,  0);
		float3 fluence21 = safeGetFlue(  i, 0,  1,  0);
		float3 fluence30 = safeGetFlue(  i, 0,  0, -1);
		float3 fluence31 = safeGetFlue(  i, 0,  0,  1);

		float3 numerator = GetDensity(i)*sigma_t*dx*dx*gridBuffer[i]+(Ds10*fluence10 + Ds11*fluence11 + Ds20*fluence20+Ds21*fluence21+Ds30*fluence30+Ds31*fluence31);
		float3 denominator = (1.f-alpha_value)*sigmaP*dx*dx+(Ds10+Ds11+Ds20+Ds21+Ds30+Ds31);
		//error_p = (numerator - $fluence[i]*denominator)/$dx/$dx
		update_fluence = numerator/denominator;
	}
	gridFluence[i] = max(weight*update_fluence+(1.f-weight)* gridFluence[i], make_float3(0.f,0.f,0.f));

}

//-----------------------------------------------------------------------------
//
//  Main program
//
//-----------------------------------------------------------------------------
RT_PROGRAM void PreCompution()
{
	//if(0)//if(curIterator>0)
	//{
	//	MultiCompution();
	//	return;
	//}

	int maxDepth = 1;
	float3 p = GetPosition( i2xyz(gridIndex));
	float3 ray_direction, ray_origin;
	Ray ray;
	//prd.seed = seed;
	float3 result = make_float3(0.0f);
	unsigned int seed = tea<16>(gridIndex*gridIndex, numCompution);
	for(int i=0; i<numSampling; ++i)
	{
		PerRayData_pathtrace prd;
		prd.attenuation = make_float3(1.f);
		prd.result = make_float3(0.f);
		prd.radiance = make_float3(0.f);
		prd.countEmitted = true;
		prd.done = false;
		prd.inside = true;
		prd.depth = 0;
		prd.seed = seed;

		float3 pdf=make_float3(1.f);
		if(1)//importance sampling
		{
			ray_direction = sampleEnvmap( pdf, rnd(prd.seed), rnd(prd.seed));
		}
		else//uniform sampling
		{
			ray_direction = uniformSphere( rnd(prd.seed), rnd(prd.seed), make_float3(1.f, 0.f, 0.f));
		}

		// [Debug] Record and plot the sampled point in environment map
		uint2 iuv = uv2iuv(make_float2(pdf));
		atomicAdd(&pixelIsSampled[iuv],1);
		// [Debug] Print the pdf value : assert( pdf nearly> 1)
		//if(gridIndex==10001)
		//{
		//	printf("%lf\n",pdf.z);
		//}

		ray_origin = p;
		//float3 envmapColor = envmapEvalLandPdf(ray_direction);//pdf.z;
		while(1)
		{
			ray = make_Ray(ray_origin, ray_direction, pathtrace_ray_type, scene_epsilon, RT_DEFAULT_MAX);
			rtTrace(top_object, ray, prd);
			if(prd.done ||(prd.depth >= maxDepth))
			{
				if(prd.done) prd.result = prd.radiance * prd.attenuation/pdf.z;
				break;
			}
			prd.depth++;
			prd.result += prd.radiance * prd.attenuation;
			ray_origin = prd.origin;
			ray_direction = prd.direction;
		}
		result += prd.result;
		seed = prd.seed;
	}
	result /= (float)numSampling;
	if (numCompution > 1)
	{
		float a = 1.0f / (float)numCompution;
		float b = ((float)numCompution - 1.0f) * a;
		float3 old_color = gridBuffer[gridIndex];
		gridBuffer[gridIndex] = a * result + b * old_color;
	}
	else
	{
		gridBuffer[gridIndex] = result;
	}
	//gridBuffer[ gridIndex] =  result/(float)numSampling;
}

//-----------------------------------------------------------------------------
//
//  Exception program
//
//-----------------------------------------------------------------------------

RT_PROGRAM void exception()
{
	gridBuffer[ gridIndex] =  make_float3(1.f,0.f,0.f);//bad_color;
}


//-----------------------------------------------------------------------------
//
//  Miss program
//
//-----------------------------------------------------------------------------




//RT_PROGRAM void envmap_miss()
//{
//	float theta = atan2f( ray.direction.x, ray.direction.z );
//	float phi   = M_PIf * 0.5f -  acosf( ray.direction.y );
//	float u     = (theta + M_PIf) * (0.5f * M_1_PIf);
//	float v     = 0.5f * ( 1.0f + sin(phi) );
//	//current_prd.radiance = bg_color*100.f;
//	current_prd.radiance = make_float3( tex2D(envmap, u, v) )*1.f;
//	
//	current_prd.done = true;
//	//current_prd.attenuation *= 0.1f;
//}
