#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include "volume.cuh"
rtDeclareVariable(uint, gridIndex, rtLaunchIndex, );
rtDeclareVariable(int, numSampling, , );
//rtBuffer<float4, 3>    gridBuffer;
static __device__ __inline__ float3 GetPosition( uint3 index)
{
	float3 p01 = P1-P0;
	return make_float3(p01.x/(float)(index_x-1)*(float)(index.x)+P0.x, p01.y/(float)(index_y-1)*(float)(index.y)+P0.y, p01.z/(float)(index_z-1)*(float)(index.z)+P0.z );
}

//-----------------------------------------------------------------------------
//
//  Main program
//
//-----------------------------------------------------------------------------
RT_PROGRAM void PreCompution()
{
	int maxDepth = 2;
	float3 p = GetPosition( i2xyz(gridIndex));
	float3 ray_direction, ray_origin;
	Ray ray;
	//prd.seed = seed;
	float3 result = make_float3(0.0f);
	unsigned int seed = tea<16>(gridIndex*gridIndex, 1);
	for(int i=0; i<numSampling; ++i)
	{
		PerRayData_pathtrace prd;
		prd.attenuation = make_float3(1.f);
		prd.result = make_float3(0.f);
		prd.radiance = make_float3(0.f);
		prd.countEmitted = true;
		prd.done = false;
		prd.inside = true;
		prd.depth = 0;
		prd.seed = seed;
		ray_direction = uniformSphere( rnd(prd.seed), rnd(prd.seed), make_float3(1.f, 0.f, 0.f));
		ray_origin = p;
		while(1)
		{
			ray = make_Ray(ray_origin, ray_direction, pathtrace_ray_type, scene_epsilon, RT_DEFAULT_MAX);
			rtTrace(top_object, ray, prd);
			if(prd.done ||(prd.depth >= maxDepth))
			{
				prd.result += prd.radiance * prd.attenuation;
				break;
			}
			prd.depth++;
			prd.result += prd.radiance * prd.attenuation;
			ray_origin = prd.origin;
			ray_direction = prd.direction;
		}
		result += prd.result;
		seed = prd.seed;
	}
	gridBuffer[ gridIndex] =  result/(float)numSampling;
}

//-----------------------------------------------------------------------------
//
//  Exception program
//
//-----------------------------------------------------------------------------

RT_PROGRAM void exception()
{
	gridBuffer[ gridIndex] =  make_float3(1.f,0.f,0.f);//bad_color;
}


//-----------------------------------------------------------------------------
//
//  Miss program
//
//-----------------------------------------------------------------------------




RT_PROGRAM void envmap_miss()
{
	float theta = atan2f( ray.direction.x, ray.direction.z );
	float phi   = M_PIf * 0.5f -  acosf( ray.direction.y );
	float u     = (theta + M_PIf) * (0.5f * M_1_PIf);
	float v     = 0.5f * ( 1.0f + sin(phi) );
	current_prd.radiance = bg_color*100.f;
	//current_prd.radiance = make_float3( tex2D(envmap, u, v) )*1.f;
	
	current_prd.done = true;
	//current_prd.attenuation *= 0.1f;
}
