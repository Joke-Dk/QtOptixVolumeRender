#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include "volume.cuh"
#include "MultiCompution.cuh"
rtDeclareVariable(uint, gridIndex, rtLaunchIndex, );
rtDeclareVariable(int, numSampling, , );
rtDeclareVariable(unsigned int,  numCompution, , );
//rtBuffer<float4, 3>    gridBuffer;
static __device__ __inline__ float3 GetPosition( int3 index)
{
	float3 p01 = P1-P0;
	return make_float3(p01.x/(float)(index_x-1)*(float)(index.x)+P0.x, p01.y/(float)(index_y-1)*(float)(index.y)+P0.y, p01.z/(float)(index_z-1)*(float)(index.z)+P0.z );
}

RT_PROGRAM void MultiCompution()
{
	if (curIterator==1)
	{
		//gridBuffer[gridIndex] *= volume_density[gridIndex]*sigma_t;
		gridFluence[ gridIndex] = make_float3(1.f)*ee*dx*J_mean;
		return;
	}
	int i = gridIndex;
	float3 update_fluence = make_float3(0.f);
	int3 xyzIndex = i2xyz( i);
	int tmpI = xyzIndex.x+xyzIndex.y+xyzIndex.z;
	if( tmpI%2 == curIterator%2) return;

	float3 sigmaP = GetSigmaT0( i);
	float3 Dp = GetDp( i);

	if ( 0)//volume_density[ i] == 0.f)
	{

	}
	else
	{
		float3 Ds10 = (safeGetDp(i, -1, 0, 0)+Dp)/2.f;
		float3 Ds11 = (safeGetDp(i,  1, 0, 0)+Dp)/2.f;
		
		float3 Ds20 = (safeGetDp(i, 0, -1, 0)+Dp)/2.f;
		float3 Ds21 = (safeGetDp(i, 0,  1, 0)+Dp)/2.f;
		
		float3 Ds30 = (safeGetDp(i, 0, 0, -1)+Dp)/2.f;
		float3 Ds31 = (safeGetDp(i, 0, 0,  1)+Dp)/2.f;
		
		float3 fluence10 = safeGetFlue(  i, -1, 0,  0);
		float3 fluence11 = safeGetFlue(  i, 1,  0,  0);
		float3 fluence20 = safeGetFlue(  i, 0, -1,  0);
		float3 fluence21 = safeGetFlue(  i, 0,  1,  0);
		float3 fluence30 = safeGetFlue(  i, 0,  0, -1);
		float3 fluence31 = safeGetFlue(  i, 0,  0,  1);

		float3 numerator = volume_density[i]*sigma_t*dx*dx*gridBuffer[i]+(Ds10*fluence10 + Ds11*fluence11 + Ds20*fluence20+Ds21*fluence21+Ds30*fluence30+Ds31*fluence31);
		float3 denominator = (1.f-alpha_value)*sigmaP*dx*dx+(Ds10+Ds11+Ds20+Ds21+Ds30+Ds31);
		//error_p = (numerator - $fluence[i]*denominator)/$dx/$dx
		update_fluence = numerator/denominator;
	}
	gridFluence[i] = max(weight*update_fluence+(1.f-weight)* gridFluence[i], make_float3(0.f,0.f,0.f));
}

//-----------------------------------------------------------------------------
//
//  Main program
//
//-----------------------------------------------------------------------------
RT_PROGRAM void PreCompution()
{
	//if(0)//if(curIterator>0)
	//{
	//	MultiCompution();
	//	return;
	//}

	int maxDepth = 1;
	float3 p = GetPosition( i2xyz(gridIndex));
	float3 ray_direction, ray_origin;
	Ray ray;
	//prd.seed = seed;
	float3 result = make_float3(0.0f);
	unsigned int seed = tea<16>(gridIndex*gridIndex, numCompution);
	for(int i=0; i<numSampling; ++i)
	{
		PerRayData_pathtrace prd;
		prd.attenuation = make_float3(1.f);
		prd.result = make_float3(0.f);
		prd.radiance = make_float3(0.f);
		prd.countEmitted = true;
		prd.done = false;
		prd.inside = true;
		prd.depth = 0;
		prd.seed = seed;
		ray_direction = uniformSphere( rnd(prd.seed), rnd(prd.seed), make_float3(1.f, 0.f, 0.f));
		ray_origin = p;
		while(1)
		{
			ray = make_Ray(ray_origin, ray_direction, pathtrace_ray_type, scene_epsilon, RT_DEFAULT_MAX);
			rtTrace(top_object, ray, prd);
			if(prd.done ||(prd.depth >= maxDepth))
			{
				prd.result += prd.radiance * prd.attenuation;
				break;
			}
			prd.depth++;
			prd.result += prd.radiance * prd.attenuation;
			ray_origin = prd.origin;
			ray_direction = prd.direction;
		}
		result += prd.result;
		seed = prd.seed;
	}
	result /= (float)numSampling;
	if (numCompution > 1)
	{
		float a = 1.0f / (float)numCompution;
		float b = ((float)numCompution - 1.0f) * a;
		float3 old_color = gridBuffer[gridIndex];
		gridBuffer[gridIndex] = a * result + b * old_color;
	}
	else
	{
		gridBuffer[gridIndex] = result;
	}
	//gridBuffer[ gridIndex] =  result/(float)numSampling;
}

//-----------------------------------------------------------------------------
//
//  Exception program
//
//-----------------------------------------------------------------------------

RT_PROGRAM void exception()
{
	gridBuffer[ gridIndex] =  make_float3(1.f,0.f,0.f);//bad_color;
}


//-----------------------------------------------------------------------------
//
//  Miss program
//
//-----------------------------------------------------------------------------




RT_PROGRAM void envmap_miss()
{
	float theta = atan2f( ray.direction.x, ray.direction.z );
	float phi   = M_PIf * 0.5f -  acosf( ray.direction.y );
	float u     = (theta + M_PIf) * (0.5f * M_1_PIf);
	float v     = 0.5f * ( 1.0f + sin(phi) );
	//current_prd.radiance = bg_color*100.f;
	current_prd.radiance = make_float3( tex2D(envmap, u, v) )*1.f;
	
	current_prd.done = true;
	//current_prd.attenuation *= 0.1f;
}
