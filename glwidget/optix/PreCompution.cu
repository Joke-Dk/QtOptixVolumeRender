#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include "volume.cuh"
rtDeclareVariable(uint3, gridIndex, rtLaunchIndex, );
rtDeclareVariable(int, numSampling, , );
rtBuffer<float4, 3>    gridBuffer;
static __device__ __inline__ float3 GetPosition( uint3 index)
{
	float3 p01 = P1-P0;
	return make_float3(p01.x/(float)(index_x-1)*(float)(index.x)+P0.x, p01.y/(float)(index_y-1)*(float)(index.y)+P0.y, p01.z/(float)(index_z-1)*(float)(index.z)+P0.z );
}

//-----------------------------------------------------------------------------
//
//  Main program
//
//-----------------------------------------------------------------------------
RT_PROGRAM void PreCompution()
{
	unsigned int seed = tea<16>(gridIndex.z*index_x*index_y + gridIndex.y*index_x + gridIndex.z, 0);
	float3 ray_origin = GetPosition( gridIndex);
	float3 ray_direction;
	Ray ray;
	PerRayData_pathtrace prd;
	prd.result = make_float3(0.f);
	prd.seed = seed;
	for(int i=0; i<numSampling; ++i)
	{
		ray_direction = uniformSphere( rnd(prd.seed), rnd(prd.seed), make_float3(1.f));
		ray = make_Ray(ray_origin, ray_direction, pathtrace_ray_type, scene_epsilon, RT_DEFAULT_MAX);
		prd.depth = 0;
		Ray ray = make_Ray(ray_origin, ray_direction, pathtrace_ray_type, scene_epsilon, RT_DEFAULT_MAX);
		rtTrace(top_object, ray, prd);
		prd.result += prd.radiance * prd.attenuation;
	}
	gridBuffer[ gridIndex] =  make_float4(prd.result / numSampling, 0.f);
}

//-----------------------------------------------------------------------------
//
//  Exception program
//
//-----------------------------------------------------------------------------

RT_PROGRAM void exception()
{
	gridBuffer[ gridIndex] =  make_float4( bad_color, 0.f);
}


//-----------------------------------------------------------------------------
//
//  Miss program
//
//-----------------------------------------------------------------------------

RT_PROGRAM void miss()
{
	current_prd.radiance = bg_color;
	current_prd.done = true;
}


rtTextureSampler<float4, 2> envmap;
RT_PROGRAM void envmap_miss()
{
	float theta = atan2f( ray.direction.x, ray.direction.z );
	float phi   = M_PIf * 0.5f -  acosf( ray.direction.y );
	float u     = (theta + M_PIf) * (0.5f * M_1_PIf);
	float v     = 0.5f * ( 1.0f + sin(phi) );
	//current_prd.radiance = bg_color;
	current_prd.radiance = make_float3( tex2D(envmap, u, v) )*1.f;
	
	current_prd.done = true;
	//current_prd.attenuation *= 0.1f;
}
