#include "hip/hip_runtime.h"
#include "volume.cuh"

rtDeclareVariable(float3,        diffuse_color, , );

RT_PROGRAM void diffuse()
{
	float3 world_shading_normal   = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
	float3 world_geometric_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );

	float3 ffnormal = faceforward( world_shading_normal, -ray.direction, world_geometric_normal );

	float3 hitpoint = ray.origin + t_hit * ray.direction;
	current_prd.origin = hitpoint;

	float z1=rnd(current_prd.seed);
	float z2=rnd(current_prd.seed);
	float3 p;
	cosine_sample_hemisphere(z1, z2, p);
	float3 v1, v2;
	createONB(ffnormal, v1, v2);
	current_prd.direction = v1 * p.x + v2 * p.y + ffnormal * p.z;
	float3 normal_color = (normalize(world_shading_normal)*0.5f + 0.5f)*0.9;
	current_prd.attenuation = current_prd.attenuation * diffuse_color*0.8f; // use the diffuse_color as the diffuse response
	current_prd.countEmitted = false;

	// Compute direct light...
	// Or shoot one...
	unsigned int num_lights = lights.size();
	float3 result = make_float3(0.0f);

	for(int i = 0; i < num_lights; ++i) 
	{
		ParallelogramLight light = lights[i];
		float z1 = rnd(current_prd.seed);
		float z2 = rnd(current_prd.seed);
		float3 light_pos = light.corner + light.v1 * z1 + light.v2 * z2;

		float Ldist = length(light_pos - hitpoint);
		float3 L = normalize(light_pos - hitpoint);
		float nDl = dot( ffnormal, L );
		float LnDl = dot( light.normal, L );
		float A = length(cross(light.v1, light.v2));

		// cast shadow ray
		if ( nDl > 0.0f && LnDl > 0.0f ) 
		{
			PerRayData_pathtrace_shadow shadow_prd;
			shadow_prd.attenuation = make_float3(1.f);
			Ray shadow_ray = make_Ray( hitpoint, L, pathtrace_shadow_ray_type, scene_epsilon, Ldist );
			//rtTrace(top_object, shadow_ray, shadow_prd);

			//dk
			float3 p=shadow_ray.origin;
			float3 dir=shadow_ray.direction;
			float l=0.f;
			float step=30.f;
			float rand0=rnd(current_prd.seed);
			l+=rand0*step;
			float tau = 0.f;
			tau+=get_density(p)*rand0;
			while (l<=300.f)
			{
				tau+=get_density(p+dir*l);
				l+=step;
			}
			tau*=step;
			shadow_prd.attenuation *=  exp(-sigma_t*tau);//make_float3(0.f);

			float3 light_attenuation = shadow_prd.attenuation;
			if(fmaxf(light_attenuation) > 0.0f)
			{
				float weight=nDl * LnDl * A / (M_PIf*Ldist*Ldist);
				result += light.emission * weight*light_attenuation;
			}
		}
	}

	current_prd.radiance = result;
}


RT_PROGRAM void shadow()
{
	current_prd_shadow.attenuation = make_float3(0.f);
	rtTerminateRay();
}