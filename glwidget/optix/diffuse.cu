#include "hip/hip_runtime.h"
//#include "volume.cuh"
#include "ray.cuh"

rtDeclareVariable(float3,        diffuse_color, , );

RT_PROGRAM void diffuse()
{
	current_prd.insertedDiffuse = true;

	float3 world_shading_normal   = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
	float3 world_geometric_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );

	float3 ffnormal = faceforward( world_shading_normal, -ray.direction, world_geometric_normal );

	float3 hitpoint = ray.origin + t_hit * ray.direction;
	current_prd.origin = hitpoint;

	float z1=rnd(current_prd.seed);
	float z2=rnd(current_prd.seed);
	float3 p;
	cosine_sample_hemisphere(z1, z2, p);
	float3 v1, v2;
	createONB(ffnormal, v1, v2);
	current_prd.direction = v1 * p.x + v2 * p.y + ffnormal * p.z;
	//float3 normal_color = (normalize(world_shading_normal)*0.5f + 0.5f)*0.9;
	current_prd.attenuation = current_prd.attenuation * diffuse_color*0.8f; // use the diffuse_color as the diffuse response

	//////////////////////////////////////////////////////////////////////////
	// Fast preview moduel
	//current_prd.attenuation *= dot(current_prd.direction, ffnormal);
	if (current_prd.depth >5)
		current_prd.done = 1;
	
	current_prd.countEmitted = false;

	// Compute direct light...
	// Or shoot one...
	unsigned int num_lights = lights.size();
	float3 result = make_float3(0.0f);

	for(int i = 0; i < num_lights; ++i) 
	{
		ParallelogramLight light = lights[i];
		float z1 = rnd(current_prd.seed);
		float z2 = rnd(current_prd.seed);
		float3 light_pos = light.corner + light.v1 * z1 + light.v2 * z2;

		float Ldist = length(light_pos - hitpoint);
		float3 L = normalize(light_pos - hitpoint);
		float nDl = dot( ffnormal, L );
		float LnDl = dot( light.normal, L );
		float A = length(cross(light.v1, light.v2));

		// cast shadow ray
		if ( nDl > 0.0f && LnDl > 0.0f ) 
		{
			PerRayData_pathtrace_shadow shadow_prd;
			shadow_prd.attenuation = make_float3(1.f);
			Ray shadow_ray = make_Ray( current_prd.origin, L, pathtrace_shadow_ray_type, scene_epsilon, Ldist );
			shadow_prd.origin = current_prd.origin;
			shadow_prd.direction = L;
			shadow_prd.seed = current_prd.seed;
			rtTrace(top_object, shadow_ray, shadow_prd);

			float3 light_attenuation = shadow_prd.attenuation;
			if(fmaxf(light_attenuation) > 0.0f)
			{
				float weight=nDl * LnDl * A / (M_PIf*Ldist*Ldist);
				result += light.emission * weight*light_attenuation;
			}
		}
	}

	current_prd.radiance = result;
}


RT_PROGRAM void shadow()
{
	current_prd_shadow.attenuation = make_float3(0.f);
	rtTerminateRay();
}