#include "hip/hip_runtime.h"
#include "ray.cuh"
// -----------------------------------------------------------------------------
//-----------------------------------------------------------------------------
//
// Glass program
//
//-----------------------------------------------------------------------------

//
// Dielectric surface shader
//
rtDeclareVariable(float3,       cutoff_color, , );
rtDeclareVariable(float,        fresnel_exponent, , );
rtDeclareVariable(float,        fresnel_minimum, , );
rtDeclareVariable(float,        fresnel_maximum, , );
rtDeclareVariable(float,        refraction_index, , );
rtDeclareVariable(int,          refraction_maxdepth, , );
rtDeclareVariable(int,          reflection_maxdepth, , );
rtDeclareVariable(float3,       refraction_color, , );
rtDeclareVariable(float3,       reflection_color, , );
rtDeclareVariable(float3,       extinction_constant, , );

rtDeclareVariable(float,    importance_cutoff, , );   
rtDeclareVariable(float3,        glass_color, , );
rtDeclareVariable(float,         index_of_refraction, , );


RT_PROGRAM void glass_fresnel()
{
  float3 world_shading_normal   = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
  float3 world_geometric_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );

  float3 ffnormal = faceforward( world_shading_normal, -ray.direction, world_geometric_normal );

  float3 hitpoint = ray.origin + t_hit * ray.direction;
  current_prd.origin = hitpoint;
  current_prd.countEmitted = false;

  float iof;
  if (current_prd.inside) {
    // Shoot outgoing ray
    iof = 1.0f/index_of_refraction;
  } else {
    iof = index_of_refraction;
  }
  float3 t;
  refract(t, ray.direction, ffnormal, iof);
  float rand_reflect = rnd(current_prd.seed);
  // check for external or internal reflection
  float cos_theta = dot(ray.direction, ffnormal);
  if (cos_theta < 0.0f)
    cos_theta = -cos_theta;
  else
    cos_theta = dot(t, ffnormal);

  float reflection = fresnel_schlick(cos_theta, fresnel_exponent, fresnel_minimum, fresnel_maximum);

  if (rand_reflect<=reflection)
  {
	 //reflect
     current_prd.direction = reflect(ray.direction, ffnormal);	
  }
  else
  {
	  //refract
	  refract(current_prd.direction, ray.direction, ffnormal, iof);
	  
	  if (current_prd.inside) {
		// Compute Beer's law
		current_prd.attenuation = current_prd.attenuation * powf(glass_color, t_hit);
	  }
	  current_prd.inside = !current_prd.inside;
  }

  current_prd.radiance = make_float3(0.0f,0.f,0.f);
}


rtDeclareVariable(float3, shadow_attenuation, , );

RT_PROGRAM void glass_any_hit_shadow()
{
  float3 world_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
  float nDi = fabs(dot(world_normal, ray.direction));

  current_prd_shadow.attenuation *= 1-fresnel_schlick(nDi, 5, 1-shadow_attenuation, make_float3(1));

  rtIgnoreIntersection();
}